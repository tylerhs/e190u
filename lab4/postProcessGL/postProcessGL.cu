#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Utilities and system includes

#include <hip/hip_runtime_api.h>

#ifndef USE_TEXTURE_RGBA8UI
texture<float4, 2, hipReadModeElementType> inTex;
#else
texture<uchar4, 2, hipReadModeElementType> inTex;
#endif

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

// get pixel from 2D image, with clamping to border
__device__ uchar4 getPixel(int x, int y)
{
#ifndef USE_TEXTURE_RGBA8UI
    float4 res = tex2D(inTex, x, y);
    uchar4 ucres = make_uchar4(res.x*255.0f, res.y*255.0f, res.z*255.0f, res.w*255.0f);
#else
    uchar4 ucres = tex2D(inTex, x, y);
#endif
    return ucres;
}

// macros to make indexing shared memory easier
#define SMEM(X, Y) sdata[(Y)*tilew+(X)]

/*
    2D convolution using shared memory
    - operates on 8-bit RGB data stored in 32-bit int
    - assumes kernel radius is less than or equal to block size
    - not optimized for performance
     _____________
    |   :     :   |
    |_ _:_____:_ _|
    |   |     |   |
    |   |     |   |
    |_ _|_____|_ _|
  r |   :     :   |
    |___:_____:___|
      r    bw   r
    <----tilew---->
*/

__global__ void
cudaProcess(unsigned int *g_odata, int imgw, int imgh,
            int tilew, int r, float threshold, float highlight, float w0, float w1, float w2, float w3, float w4, float w5, float w6, float w7, float w8)
{
    extern __shared__ uchar4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

#if 0
    uchar4 c4 = getPixel(x, y);
    g_odata[y*imgw+x] = rgbToInt(c4.z, c4.y, c4.x);
#else
    // copy tile to shared memory
    // center region
    SMEM(r + tx, r + ty) = getPixel(x, y);

    // borders
    if (threadIdx.x < r)
    {
        // left
        SMEM(tx, r + ty) = getPixel(x - r, y);
        // right
        SMEM(r + bw + tx, r + ty) = getPixel(x + bw, y);
    }

    if (threadIdx.y < r)
    {
        // top
        SMEM(r + tx, ty) = getPixel(x, y - r);
        // bottom
        SMEM(r + tx, r + bh + ty) = getPixel(x, y + bh);
    }

    // load corners
    if ((threadIdx.x < r) && (threadIdx.y < r))
    {
        // tl
        SMEM(tx, ty) = getPixel(x - r, y - r);
        // bl
        SMEM(tx, r + bh + ty) = getPixel(x - r, y + bh);
        // tr
        SMEM(r + bw + tx, ty) = getPixel(x + bh, y - r);
        // br
        SMEM(r + bw + tx, r + bh + ty) = getPixel(x + bw, y + bh);
    }

    // wait for loads to complete
    __syncthreads();

    // perform convolution
    float rsum = 0.0f;
    float gsum = 0.0f;
    float bsum = 0.0f;
    float samples = 0.0f;
    float weightSum = 0.0f;
    
    float convMat[9] = {w0,w1,w2,w3,w4,w5,w6,w7,w8};
    for(int i = 0; i < 9; i++)
		weightSum += convMat[i];

    for (int dy=-r; dy<=r; dy++)
    {
        for (int dx=-r; dx<=r; dx++)
        {
#if 0
            // try this to see the benefit of using shared memory
            uchar4 pixel = getPixel(x+dx, y+dy);
#else
            uchar4 pixel = SMEM(r+tx+dx, r+ty+dy);
#endif

            // only sum pixels within disc-shaped kernel
            //float l = dx*dx + dy*dy;
            //if (l <= r*r)
            //{
				int index = (dx+1)+(dy+1)*3;
                float r = float(pixel.x);//*convMat[index];
                float g = float(pixel.y);//*convMat[index];
                float b = float(pixel.z);//*convMat[index];
#if 1
				
				
				
                r *= convMat[index];
                g *= convMat[index];
                b *= convMat[index];

#endif

				rsum += r;
				gsum += g;
				bsum += b;
                samples += 1.0f;
            //}
        }
    }

    //rsum /= weightSum/4;
    //gsum /= weightSum/4;
    //bsum /= weightSum/4;
    // ABGR
    g_odata[y*imgw+x] = rgbToInt(rsum, gsum, bsum);
    //g_odata[y*imgw+x] = rgbToInt(x,y,0);
#endif
}

extern "C" void
launch_cudaProcess(dim3 grid, dim3 block, int sbytes,
                   hipArray *g_data_array, unsigned int *g_odata,
                   int imgw, int imgh, int tilew,
                   int radius, float threshold, float highlight, float convMat[])
{
    checkCudaErrors(hipBindTextureToArray(inTex, g_data_array));

    struct hipChannelFormatDesc desc;
    checkCudaErrors(hipGetChannelDesc(&desc, g_data_array));

#if 0
    printf("CUDA Array channel descriptor, bits per component:\n");
    printf("X %d Y %d Z %d W %d, kind %d\n",
           desc.x,desc.y,desc.z,desc.w,desc.f);

    printf("Possible values for channel format kind: i %d, u%d, f%d:\n",
           hipChannelFormatKindSigned, hipChannelFormatKindUnsigned,
           hipChannelFormatKindFloat);
#endif
	//float convMat[9];
	//for(int count = 0; count < 9; count++){
	//	convMat[count] = 1.0;
	//}
	float w0 = convMat[0];
	float w1 = convMat[1];
	float w2 = convMat[2];
	float w3 = convMat[3];
	float w4 = convMat[4];
	float w5 = convMat[5];
	float w6 = convMat[6];
	float w7 = convMat[7];
	float w8 = convMat[8];

    //printf("\n");
#ifdef GPU_PROFILING
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);

    int nIter = 30;

    for (int i = -1; i < nIter; ++i)
    {
        if (i == 0)
        {
            sdkStartTimer(&timer);
        }

#endif

        cudaProcess<<< grid, block, sbytes >>>(g_odata, imgw, imgh,
                                               block.x+(2*radius), radius, 0.8f, 4.0f, w0,w1,w2,w3,w4,w5,w6,w7,w8);

#ifdef GPU_PROFILING
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    double dSeconds = sdkGetTimerValue(&timer)/((double)nIter * 1000.0);
    double dNumTexels = (double)imgw * (double)imgh;
    double mtexps = 1.0e-6 * dNumTexels/dSeconds;

    if (radius == 4)
    {
        printf("\n");
        printf("postprocessGL, Throughput = %.4f MTexels/s, Time = %.5f s, Size = %.0f Texels, NumDevsUsed = %d, Workgroup = %u\n",
               mtexps, dSeconds, dNumTexels, 1, block.x * block.y);
    }

#endif
}
